
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
extern "C" 
void vecadd_cuda(double *h_a, double *h_b, double *h_c, int *length)
{
    // Size of vectors
    int n = *length;
 
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 

    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 500;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}