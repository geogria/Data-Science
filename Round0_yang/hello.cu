
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)
//GPU function, execute on GPU(device)
 __global__ void myfunction(void) 
{

}

//standard C function, execute on CPU(host)
int main(void) 
{
	int blockSize = 1, gridSize =1;
	myfunction<<<gridSize, blockSize>>>();
	cudaCheckErrors("hello world fail");
	printf("Hello World!\n");
	return 0;
}